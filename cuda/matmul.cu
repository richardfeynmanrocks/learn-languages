#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <eigen3/Eigen/Dense>
// Kernel function to add the elements of two arrays
__global__
void add(int n, Eigen::MatrixXf *x, Eigen::MatrixXf *y, Eigen::MatrixXf *z)
{
    int index = blockIdx.x * + blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  
    for (int i = index; i < n; i+=stride)
        z[i] = x[i] * y[i];
}

int main(void)
{
    int N = 10000;
    Eigen::MatrixXf *x, *y, *z, *z2;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged((void**)&x, N*sizeof(Eigen::MatrixXf));
    hipMallocManaged((void**)&y, N*sizeof(Eigen::MatrixXf));
    hipMallocManaged((void**)&z, N*sizeof(Eigen::MatrixXf));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = Eigen::MatrixXf::Random(100,100);
        y[i] = Eigen::MatrixXf::Random(100,100);
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    // Run kernel on 1M elements on the GPU
    add<<<numBlocks, blockSize>>>(N, x, y, z);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Free memory
    hipFree(x);
    hipFree(y);
  
    return 0;
}
